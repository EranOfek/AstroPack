#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <iomanip>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace std;

__global__ void cuda_NDFT_gpu(const double* t, const double* m, const double* f,
    double* iter_result_real, double* iter_result_imag, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) {
        double sum_real = 0.0;
        double sum_imag = 0.0;

        for (int j = 0; j < N; ++j) {
            sum_real += m[j] * cos(-2 * M_PI * f[tid] * t[j]);
            sum_imag += m[j] * sin(-2 * M_PI * f[tid] * t[j]);
        }

        iter_result_real[tid] = sum_real / N;
        iter_result_imag[tid] = sum_imag / N;
    }
}

int main() {

    int NN[4] = { 10, 100, 1000, 10000 };
    int iters = 100;

    for (int iii = 0; iii < 4; ++iii) {
        double time_cuda = 0.0;

        int N = NN[iii];

        double* t = new double[N];
        double* m = new double[N];
        double* f = new double[N];

        double* result_real = new double[N];
        double* result_imag = new double[N];

        // Generate file name dynamically
        std::stringstream filename;
        filename << "../inputs/ndft_input_" << N << ".csv";
        std::ifstream file(filename.str());

        if (!file.is_open()) {
            cout << "Error opening file: " << filename.str() << endl;
            return 1; // exit with an error code
        }

        std::string line;

        for (int i = 0; i < 3; ++i) {
            std::getline(file, line);
            std::istringstream iss(line);

            for (int j = 0; j < N; ++j) {
                std::string valueStr;
                std::getline(iss, valueStr, ',');

                // Convert the string to a double
                double value = std::stod(valueStr);

                if (i == 0) {
                    t[j] = value;
                }
                else if (i == 1) {
                    m[j] = value;
                }
                else if (i == 2) {
                    f[j] = value;
                }
            }
        }

        // Define grid and block sizes
        int blockSize = 16;
        int numBlocks = (N + blockSize - 1) / blockSize;

        for (int iter = 0; iter < iters; ++iter) {

            auto ttic = std::chrono::high_resolution_clock::now();

            double* iter_result_real = new double[N];
            double* iter_result_imag = new double[N];

            // Allocate memory on the GPU
            double* d_t, * d_m, * d_f, * d_iter_result_real, * d_iter_result_imag;
            hipMalloc(&d_t, N * sizeof(double));
            hipMalloc(&d_m, N * sizeof(double));
            hipMalloc(&d_f, N * sizeof(double));
            hipMalloc(&d_iter_result_real, N * sizeof(double));
            hipMalloc(&d_iter_result_imag, N * sizeof(double));

            // Copy data from host to device
            hipMemcpy(d_t, t, N * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_m, m, N * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_f, f, N * sizeof(double), hipMemcpyHostToDevice);

            cuda_NDFT_gpu << <numBlocks, blockSize >> > (d_t, d_m, d_f, d_iter_result_real, d_iter_result_imag, N);
            hipDeviceSynchronize();

            hipMemcpy(iter_result_real, d_iter_result_real, N * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(iter_result_imag, d_iter_result_imag, N * sizeof(double), hipMemcpyDeviceToHost);

            auto elapsed_time = std::chrono::high_resolution_clock::now() - ttic;
            time_cuda += std::chrono::duration_cast<std::chrono::microseconds>(elapsed_time).count() / 1e6;

            hipFree(d_t);
            hipFree(d_m);
            hipFree(d_f);
            hipFree(d_iter_result_real);
            hipFree(d_iter_result_imag);

            // Copy values to result arrays
            std::copy(iter_result_real, iter_result_real + N, result_real);
            std::copy(iter_result_imag, iter_result_imag + N, result_imag);

            delete[] iter_result_real;
            delete[] iter_result_imag;
        }

        double time_cuda_avg = time_cuda / iters;

        cout << "GPU :: For N=" << N << ", CUDA time=" << scientific << setprecision(6) <<  time_cuda_avg << endl;

        // Clean up
        delete[] t;
        delete[] m;
        delete[] f;

        // Write result to CSV file
        std::stringstream outputFilename;
        outputFilename << "ndft_cpp_gpu_output_" << N << ".csv";
        std::ofstream outputFile(outputFilename.str());


	if (outputFile.is_open()) {
        // Write complex numbers to the file
	        for (int i = 0; i < N; ++i) {
        	    outputFile << result_real[i];
		    
		    if (result_imag[i] >= 0) {
		    	outputFile << "+";		
		    } else {
			outputFile << "-";
		    }

		   outputFile << std::abs(result_imag[i]) << "i\n";
		
	        }

        	// Close the file
        	outputFile.close();
        }

        delete[] result_real;
        delete[] result_imag;


        std::string tfilename = "output_times.csv";

        if (iii == 0) {

                std::ofstream outFile(tfilename);
                if (!outFile.is_open()) {
                        std::cerr << "Error opening file for writing headers" << std::endl;
                        return 1; // Return an error code
                }

                outFile << "N, time" << std::endl;

        }

        std::ofstream outFile(tfilename, std::ios_base::app);
        if (!outFile.is_open()) {
                std::cerr << "Error opening file for appending values" << std::endl;
                return 1; // Return an error code
        }

        outFile << N << ", " << std::scientific << std::setprecision(6) << time_cuda_avg << std::endl;


    }

    return 0;
}

