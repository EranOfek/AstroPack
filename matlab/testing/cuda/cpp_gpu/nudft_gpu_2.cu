#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cmath>
#include <chrono>
#include <fstream>
#include <sstream>
#include <stdio.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

using namespace std;

__global__ void cuda_NDFT_gpu_2(const double* t, const double* m, const double* f,
    double* freq_result_real, double* freq_result_imag, int i, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) {
        freq_result_real[tid] = m[tid] * cos(-2 * M_PI * f[i] * t[tid]);
        freq_result_imag[tid] = m[tid] * sin(-2 * M_PI * f[i] * t[tid]);
    }
}

int main() {

    int NN[4] = { 10, 100, 1000, 10000 };
    int iters = 10;

    for (int iii = 0; iii < 4; ++iii) {
        double time_cuda = 0.0;

        int N = NN[iii];

        double* t = new double[N];
        double* m = new double[N];
        double* f = new double[N];

        double* result_real = new double[N];
        double* result_imag = new double[N];

        // Generate file name dynamically
        std::stringstream filename;
        filename << "ndft_input_" << N << ".csv";
        std::ifstream file(filename.str());

        if (!file.is_open()) {
            cout << "Error opening file: " << filename.str() << endl;
            return 1; // exit with an error code
        }

        std::string line;

        for (int i = 0; i < 3; ++i) {
            std::getline(file, line);
            std::istringstream iss(line);

            for (int j = 0; j < N; ++j) {
                std::string valueStr;
                std::getline(iss, valueStr, ',');

                // Convert the string to a double
                double value = std::stod(valueStr);

                if (i == 0) {
                    t[j] = value;
                }
                else if (i == 1) {
                    m[j] = value;
                }
                else if (i == 2) {
                    f[j] = value;
                }
            }
        }

        // Define grid and block sizes
        int blockSize = 256;
        int numBlocks = (N + blockSize - 1) / blockSize;

        for (int iter = 0; iter < iters; ++iter) {

            auto ttic = std::chrono::high_resolution_clock::now();

            double* iter_result_real = new double[N];
            double* iter_result_imag = new double[N];

            // Allocate memory on the GPU
            double* d_t, * d_m, * d_f;
            hipMalloc(&d_t, N * sizeof(double));
            hipMalloc(&d_m, N * sizeof(double));
            hipMalloc(&d_f, N * sizeof(double));

            // Copy data from host to device
            hipMemcpy(d_t, t, N * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_m, m, N * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_f, f, N * sizeof(double), hipMemcpyHostToDevice);
            
            for (int i = 0; i < N; ++i) {

                double* freq_result_real = new double[N];
                double* freq_result_imag = new double[N];

                double* d_freq_result_real, * d_freq_result_imag;

                hipMalloc(&d_freq_result_real, N * sizeof(double));
                hipMalloc(&d_freq_result_imag, N * sizeof(double));

                double sum_real = 0.0;
                double sum_imag = 0.0;

                cuda_NDFT_gpu_2 << <numBlocks, blockSize >> > (d_t, d_m, d_f, d_freq_result_real, d_freq_result_imag, i, N);
                hipDeviceSynchronize();

                hipMemcpy(freq_result_real, d_freq_result_real, N * sizeof(double), hipMemcpyDeviceToHost);
                hipMemcpy(freq_result_imag, d_freq_result_imag, N * sizeof(double), hipMemcpyDeviceToHost);

                for (int j = 0; j < N; ++j) {
                    sum_real += freq_result_real[j];
                    sum_imag += freq_result_imag[j];
                }

                iter_result_real[i] = sum_real / N;
                iter_result_imag[i] = sum_imag / N;

                auto elapsed_time = std::chrono::high_resolution_clock::now() - ttic;
                time_cuda += std::chrono::duration_cast<std::chrono::microseconds>(elapsed_time).count() / 1e6;

                delete[] freq_result_real;
                delete[] freq_result_imag;

                hipFree(d_freq_result_real);
                hipFree(d_freq_result_imag);
            }

            hipFree(d_t);
            hipFree(d_m);
            hipFree(d_f);

            // Copy values to result arrays
            std::copy(iter_result_real, iter_result_real + N, result_real);
            std::copy(iter_result_imag, iter_result_imag + N, result_imag);

            delete[] iter_result_real;
            delete[] iter_result_imag;

        }

        double time_cuda_avg = time_cuda / iters;

        cout << "GPU :: For N=" << N << ", CUDA time=" << time_cuda_avg << endl;

        // Clean up
        delete[] t;
        delete[] m;
        delete[] f;

        // Write result to CSV file
        std::stringstream outputFilename;
        outputFilename << "ndft_cpp_cpu_output_" << N << ".csv";
        std::ofstream outputFile(outputFilename.str());

        for (int i = 0; i < N; ++i) {
            outputFile << result_real[i] << " + " << result_imag[i] << "i \n";
        }

        delete[] result_real;
        delete[] result_imag;

    }

    return 0;
}

